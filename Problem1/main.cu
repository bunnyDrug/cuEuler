#include <iostream>
#include <hip/hip_runtime.h>
#include <numeric>

using namespace std;

#define LIMIT 1000

__global__ void gpuFactor(int *x) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid == 0 || tid > LIMIT) { return; }

    if (tid % 3 == 0 || tid % 5 == 0) {
        x[tid] = tid;
    }
}

int main() {

    // allocate memory on the GPU
    int *device_a;
    hipMalloc((void **) &device_a, LIMIT * sizeof(int));

    // run kernel
    gpuFactor<<<5, 256>>>(device_a);

    // Copy memory from GPU back to CPU and store it in host_a
    int host_a[LIMIT];
    hipMemcpy(host_a, device_a, LIMIT * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(device_a);

    // Run accumulate sum on CPU
    cout << accumulate(host_a, host_a + LIMIT, 0) << endl;

    // TODO parallel reduce on GPU.
}
