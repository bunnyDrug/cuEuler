#include <iostream>
#include <hip/hip_runtime.h>
#include <numeric>

using namespace std;

__global__
void gpuFactor(uint *x) {
    uint tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid == 0) { return; }

    if (tid % 3 == 0 || tid % 5 == 0) {
        x[tid] = tid;
    }
}

int main() {

    int LIMIT = 1000;

    // allocate memory on the GPU
    uint *device_a;
    hipMalloc((void **) &device_a, LIMIT * sizeof(int));

    // run kernel
    gpuFactor<<<5, 200>>>(device_a);

    // Copy memory from GPU back to CPU and store it in host_a
    int host_a[LIMIT];
    hipMemcpy(host_a, device_a, LIMIT * sizeof(int), hipMemcpyDeviceToHost);

    // Run accumulate sum on CPU
    // TODO parallel reduce on GPU.
    cout << accumulate(host_a, host_a + LIMIT, 0) << endl;

}
