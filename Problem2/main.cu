#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

#define phi 1.61803398

// I do not think this solution can be done more efficiently on the GPU
int main() {

    // Calculate golden ratio to the power of three
    double phi3 = pow(phi, 3);

    int accumulator = 0;
    // every third number in the sequence is even.
    // we calculate the new sequence number and set it as the index (rounding to the nearest whole number)
    for (int i = 2; i < 4000000; i = round(i * phi3)) {
        accumulator += i;
    }

    cout << "Sum of all even Fibonacci values under 4 million = " << accumulator << endl;
}
