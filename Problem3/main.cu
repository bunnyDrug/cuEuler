#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

#define N 10

__global__ void createSieve(int* x) {
    int tid = threadIdx.x;

    int start = 2;

sieve:
    if (x[tid] != 0 && x[tid] % start == 0) {
        x[tid] = 0;
    }
    hipDeviceSynchronize();
    for (int i = 0; i < N; i ++) {
        if (x[i] > start && x[i] != 0) {
            start = x[i];
            goto sieve;
        }
    }
}

int main() {
    // allocate memory on the GPU
    int *device_a;
    hipMalloc((void **) &device_a, N * sizeof(int));

    // run kernel
    createSieve<<<1, 10>>>(device_a);

    // Copy memory from GPU back to CPU and store it in host_a
    int host_a[N];
    hipMemcpy(host_a, device_a, N * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(device_a);

    cout << host_a << endl;
}
