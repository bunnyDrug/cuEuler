#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>


void printPrimes(double *host_a, double size);

using namespace std;

__global__ void createSieve(double *x, double N) {
    for (int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < N; tid += blockDim.x * gridDim.x) {


        // we are not interested in the 0 and 1 as they are not prime
        if (tid == 1 || tid == 0) {
            continue;
        } else {

            // let's begin at 2
            double sieve = 2;

            x[tid] = tid;

            // Sieve all 2's
            if (x[tid] != 0 && fmod(x[tid], sieve) == 0 && x[tid] != sieve) {
                x[tid] = 0;
            }

            while (true) {
                if (tid > sieve && x[tid] != 0) {
                    sieve++;

                    if (x[tid] != 0 && fmod(x[tid], sieve) == 0 && x[tid] != sieve) {
                        x[tid] = 0;
                        break;
                    }
                } else {
                    break;
                }

            }
        }
    }
}

int main() {
    double limit = 600851475143;

    // allocate memory on the GPU
    double *device_a;
    hipMalloc((void **) &device_a, limit * sizeof(double));

    // run kernel
    createSieve<<<1, 1>>>(device_a, limit);

    // Copy memory from GPU back to CPU and store it in host_a
    double *host_a;
    hipMemcpy(host_a, device_a, limit * sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(device_a);


    double halfwayPoint = sqrt(limit);

    cout << "sqrt was: " << halfwayPoint << endl;
    cout << "There can be at most one other prime factor greater than " << halfwayPoint << endl;

    bool neverFound = true;
    for (int i = ceil(halfwayPoint); i < limit; i++) {
        if (host_a[i] != 0 && fmod(limit, host_a[i]) == 0) {
            cout << "Highest Prime is " << i << endl;
            neverFound = false;
            break;
        }
    }
    if (neverFound) {
        for (int i = ceil(halfwayPoint); i < limit; i--) {
            if (host_a[i] != 0 && fmod(limit, host_a[i]) == 0) {
                cout << "Highest Prime is " << i << endl;
                break;
            }
        }
    }
}

